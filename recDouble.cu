#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <sys/time.h>

#define SIZE 128
#define BLOCK_SIZE 128

double get_clock(){
        struct timeval tv;
        int ok = gettimeofday(&tv, (void *) 0);
                if (ok<0){
                        printf("gettimeofday error");
                }
                return (tv.tv_sec * 1.0 + tv.tv_usec * 1.0E-6);
        }

__global__ void scan (int * input, int * output){
        int gIndex = threadIdx.x + blockIdx.x * blockDim.x;

        if(gIndex >= SIZE){ // invalid array index
                return;
        }

        int * start;
        int * end;

        start = &input[0];
        end = &output[0];

        int* temp;

        for (int j=1; j <= SIZE; j*=2){ //add value at j to previous value
                __syncthreads();
                if(gIndex < j){
                        end[gIndex] = start[gIndex];
                }
                else{
                        end[gIndex] = input[gIndex] + start[gIndex-1];
                }

                temp = end;
                end = start;
                start = temp;
        }
        output[gIndex] = start[gIndex];
}
int main(void){
        double t0 = get_clock();

        int *input;
        int * output;
        int x;

        hipMallocManaged(&input, SIZE*sizeof(int));
        hipMallocManaged(&output, SIZE*sizeof(int));

        for(int i=0; i < SIZE; i++){
                input[i] = 1;
        }

        //number of blocks needed
        if (SIZE%BLOCK_SIZE == 0){
                x = SIZE/BLOCK_SIZE;
        }
        else {
                x = SIZE/BLOCK_SIZE + 1;
        }
        printf("Number of blocks: %d\n", x);

        //launch kernal
        scan<<<x, BLOCK_SIZE>>>(input, output);
        hipDeviceSynchronize();

        # for(int i = 0; i<SIZE; i++){
        #       printf("%d ", output[i]);
        # }
        # printf("\n");

        // print our error
        printf("%s\n", hipGetErrorString(hipGetLastError()));

        //free memory
        hipFree(input);
        hipFree(output);

        double t1 = get_clock();
        printf("time per call %f s\n", (t1-t0));

        return 0;
}
