
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <sys/time.h>

#define SIZE 128
#define BLOCK_SIZE 128

double get_clock(){
        struct timeval tv;
        int ok = gettimeofday(&tv, (void *) 0);
        if (ok < 0){
                printf("gettimeofday error");
        }
        return (tv.tv_sec * 1.0 + tv.tv_usec * 1.0E-6);
}

__global__ void scan(int* input, int* output){
        int gIndex = threadIdx.x + blockIdx.x * blockDim.x;

        if (gIndex >= SIZE){ // stop when there's no correspondng index value in the array
                return;
        }

        for (int i = 0; i<SIZE; i++){
                int value = 0;
                for (int j = 0; j <= i; j++){
                        value += input[j];
                }
                output[i] = value;
        }

        __syncthreads();

}
int main(void ){
        double t0 = get_clock();

        int *input;
        int *output;
        int x;
        hipMallocManaged(&input, SIZE*sizeof(int));
        hipMallocManaged(&output, SIZE*sizeof(int));

        for(int i = 0; i < SIZE; i++){
                input[i] = 1;
        }

        // determine number of blocks
        if (SIZE % BLOCK_SIZE == 0){
                x = SIZE/BLOCK_SIZE;
        }
        else{
                x = (SIZE/BLOCK_SIZE) + 1;
        }

        //launch kernal
        scan<<<x, BLOCK_SIZE>>>(input, output);

        hipDeviceSynchronize();

        for (int i = 0; i <SIZE; i++){
                printf("%d ", output[i]);
        }

        hipFree(input);
        hipFree(output);

        double t1 = get_clock();
        printf("time per call: %f s\n", (t1-t0));

        return 0;

}
